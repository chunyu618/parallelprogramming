#include <stdio.h>
#include <hip/hip_runtime.h>

#ifndef Size
#define Size 10
#endif

__global__ void hello(int A[Size][Size], int B[Size][Size], int C[Size][Size])
{
  int sum = 0;
  for (int k = 0; k < Size; k++)
    sum += A[threadIdx.x][k] * B[k][threadIdx.y];
  C[threadIdx.x][threadIdx.y] = sum;
}

int main(void)
{
  int *device_A, *device_B, *device_C;
  int *host_A, *host_B, *host_C;
  int size = sizeof(int) * Size * Size;
  int *aptr, *bptr;

  dim3 blocks(Size, Size);

  hipMalloc((void **)&device_A, size);
  hipMalloc((void **)&device_B, size);
  hipMalloc((void **)&device_C, size);
  host_A = (int *)malloc(size);
  host_B = (int *)malloc(size);
  host_C = (int *)malloc(size);

  aptr = host_A;
  bptr = host_B;
  for (int i = 0; i < Size; i++)
    for (int j = 0; j < Size; j++) {
      *aptr++ = *bptr++ = (i == j);
    }

  hipMemcpy(device_A, host_A, size, hipMemcpyHostToDevice);
  hipMemcpy(device_B, host_B, size, hipMemcpyHostToDevice);

  hello <<< 1, blocks >>> ((int (*)[Size])device_A, (int (*)[Size])device_B,
			   (int (*)[Size])device_C);
  hipMemcpy(host_C, device_C, size, hipMemcpyDeviceToHost);

  int k = 0;
  for (int i = 0; i < Size; i++)
    for (int j = 0; j < Size; j++)
      printf("host_C[%d][%d] = %d\n", i, j, host_C[k++]);

  hipFree(device_A);
  hipFree(device_B);
  hipFree(device_C);
  free(host_A);
  free(host_B);
  free(host_C);
}
			
