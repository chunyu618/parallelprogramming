#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>

#ifndef N
#define N 256
#endif

#define b 16

#define CHECKC

__global__ void matrixMul(int A[N][N], int B[N][N], int C[N][N])
{
  int row = blockIdx.x * b + threadIdx.x;
  int column = blockIdx.y * b + threadIdx.y;
  int sum = 0;
  for (int i = 0; i < N; i++) {
      sum += A[row][i] * B[column][i];
  }
  C[row][column] = sum;
}

int host_A[N][N], host_B[N][N], host_C[N][N];

int main(void)
{
  int *device_A, *device_B, *device_C;
  int size = sizeof(int) * N * N;
  hipMalloc((void **)&device_A, size);
  hipMalloc((void **)&device_B, size);
  hipMalloc((void **)&device_C, size);

  for (int i = 0; i < N; i++)
    for (int j = 0; j < N; j++) {
      host_A[i][j] = host_B[i][j] = ((i == j)? 1 : 0);
    }

  hipMemcpy(device_A, host_A, size, hipMemcpyHostToDevice);
  hipMemcpy(device_B, host_B, size, hipMemcpyHostToDevice);

  dim3 block(b, b);
  dim3 grid(N / b, N / b);

  float time;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  matrixMul <<< grid, block >>> ((int (*)[N])device_A, (int (*)[N])device_B,
			   (int (*)[N])device_C);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  printf("the multiplcaition takes %f seconds\n", time);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipMemcpy(host_C, device_C, size, hipMemcpyDeviceToHost);
#ifdef CHECKC
  for (int i = 0; i < N; i++)
    for (int j = 0; j < N; j++) 
      assert(host_C[i][j] == (i == j)? 1 : 0);
#endif

  hipFree(device_A);
  hipFree(device_B);
  hipFree(device_C);
}
			
