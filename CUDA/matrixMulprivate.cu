#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#ifndef N
#define N 32
#endif

#define b 8

#define PRINTC
#define CHECKC

__global__ void matrixMul(int A[N][N], int B[N][N], int C[N][N])
{
  int row = blockIdx.x * b + threadIdx.x;
  int column = blockIdx.y * b + threadIdx.y;
  __shared__ int sA[b][b];
  __shared__ int sB[b][b];

  int sum = 0;
  for (int r = 0; r < N / b; r++) {
    sA[threadIdx.x][threadIdx.y] = A[row][r * b + threadIdx.y];
    sB[threadIdx.x][threadIdx.y] = B[r * b + threadIdx.x][column];
    __syncthreads();
    for (int k = 0; k < b; k++)
      sum += sA[threadIdx.x][k] * sB[k][threadIdx.y];
    __syncthreads();
  }
  C[row][column] = sum;
}


int main(void)
{
  int *device_A, *device_B, *device_C;
  int *host_A, *host_B, *host_C;
  int size = sizeof(int) * N * N;
  int *aptr, *bptr;

  hipMalloc((void **)&device_A, size);
  hipMalloc((void **)&device_B, size);
  hipMalloc((void **)&device_C, size);
  host_A = (int *)malloc(size);
  host_B = (int *)malloc(size);
  host_C = (int *)malloc(size);

  aptr = host_A;
  bptr = host_B;
  for (int i = 0; i < N; i++)
    for (int j = 0; j < N; j++) {
      *aptr++ = *bptr++ = ((i == j)? 1 : 0);
    }

  hipMemcpy(device_A, host_A, size, hipMemcpyHostToDevice);
  hipMemcpy(device_B, host_B, size, hipMemcpyHostToDevice);

  dim3 block(b, b);
  dim3 grid(N / b, N / b);
  matrixMul <<< grid, block >>> ((int (*)[N])device_A, (int (*)[N])device_B,
			   (int (*)[N])device_C);
  hipMemcpy(host_C, device_C, size, hipMemcpyDeviceToHost);

  int k;
#ifdef PRINTC
  k = 0;
  for (int i = 0; i < N; i++)
    for (int j = 0; j < N; j++)
      printf("host_C[%d][%d] = %d\n", i, j, host_C[k++]);
#endif
#ifdef CHECKC
  k = 0;
  for (int i = 0; i < N; i++)
    for (int j = 0; j < N; j++) 
      assert(host_C[k++] == ((i == j)? 1 : 0));
#endif

  hipFree(device_A);
  hipFree(device_B);
  hipFree(device_C);
  free(host_A);
  free(host_B);
  free(host_C);
}
			
