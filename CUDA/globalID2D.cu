#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 4
__global__ void hello(int int_array[N][N])
{
  int_array[threadIdx.x][threadIdx.y]
    *= (threadIdx.x + threadIdx.y);
}

int main(void)
{
  int *device_int_array;
  int size = sizeof(int) * N * N;
  int host_int_array[N][N];
  dim3 blocks (N,N);
  hipMalloc((void **)&device_int_array, size);
  for (int i = 0; i < N; i++)
    for (int j = 0; j < N; j++)
      host_int_array[i][j] = i + j;
  hipMemcpy(device_int_array, host_int_array, size,
	     hipMemcpyHostToDevice);
  hello <<< 1, blocks >>> ((int (*)[N])device_int_array);
  hipMemcpy(host_int_array, device_int_array, size,
	     hipMemcpyDeviceToHost);
  for (int i = 0; i < N; i++)
    for (int j = 0; j < N; j++)
      printf("host_int_array[%d][%d] = %d\n", i, j,
	     host_int_array[i][j]);
  hipFree(device_int_array);
}
