#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 4
__global__ void hello(int int_array[4][N][N])
{
  int_array[blockIdx.x][threadIdx.x][threadIdx.y]
    *= ((threadIdx.x + threadIdx.y) * blockIdx.x);
}

int main(void)
{
  int *device_int_array;
  int size = sizeof(int) * N * N * 4;
  int host_int_array[4][N][N];
  dim3 blocks (N, N);
  hipMalloc((void **)&device_int_array, size);
  for (int i = 0; i < N; i++)
    for (int j = 0; j < N; j++)
      for (int k = 0; k < 4; k++)
	host_int_array[k][i][j] = i + j;
  hipMemcpy(device_int_array, host_int_array, size,
	     hipMemcpyHostToDevice);
  hello <<< 4, blocks >>> ((int (*)[N][N])device_int_array);
  hipMemcpy(host_int_array, device_int_array, size,
	     hipMemcpyDeviceToHost);
  for (int k = 0; k < 4; k++)
    for (int i = 0; i < N; i++)
      for (int j = 0; j < N; j++)
	printf("host_int_array[%d][%d][%d] = %d\n", k, i, j,
	       host_int_array[k][i][j]);
  hipFree(device_int_array);
}
